#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <cmath>
#include <fstream>
#include "FreeStreamSolver.hpp"
#include "PoissonSolver1D.hpp"
#include "Timer.h"
#include "MemSaveReorderCopy.hpp"
#include "DensityReducer.hpp"
#include "PhaseSpaceInitialization.hpp"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

using Real = float;
using Complex = std::complex<Real>;

constexpr std::size_t nx1 = 500;
constexpr std::size_t nv1 = 256;
constexpr std::size_t nx1Ghost = 6;
constexpr std::size_t nx1Tot = nx1Ghost*2+nx1;
constexpr std::size_t nTot = nx1Tot*nv1;

constexpr Real x1Max =  20;
constexpr Real x1Min =  0;
constexpr Real v1Max =  6;
constexpr Real v1Min = -6;

constexpr Real dt = (x1Max-x1Min)/nx1/v1Max/2.3;



int main(int argc, char* argv[]) {
	hipSetDevice(1);
	std::cout << "dt=" << dt << std::endl;
	Timer timer;
	
	quakins::CoordinateSystem<Real,2>
					_coord({nx1,nv1}, {nx1Ghost,0},
								 {x1Min,x1Max, v1Min,v1Max});

	auto f = [](std::array<Real,2> z) -> Real {

		auto fx = [](Real x1) {
			return 1.+.1*std::cos(2.*M_PI/x1Max*x1);
		};
		auto fv = [](Real v1) {
			return std::exp(-std::pow(v1,2)/2.)/std::sqrt(2.*M_PI);
		};

		return static_cast<Real>(fx(z[0])*fv(z[1]));
	};


	quakins::fbm::FreeStreamSolver<Real,2,0> 
					fbmSolverX1(_coord,dt*.5);	
	
	thrust::device_vector<Real> 
		ion(nTot), ion_buf(nTot),
		electron(nTot), electron_buf(nTot);
	
	timer.tick("Phase space initialization...")
	quakins::PhaseSpaceInitialization<Real,2> init(&_coord);
	init(electron.begin(),f);
	timer.tock();

	std::ofstream bout("dfbegin",std::ios::out);
	bout << electron << std::endl;


	thrust::device_vector<Real> 
		dens_e(nx1Tot), dens_i(nx1Tot), potential(nx1Tot);

	quakins::DensityReducer<Real,nv1,nx1Tot,
					thrust::device_vector> cal_dens(v1Min,v1Max);

	quakins::FFTPoissonSolver1D<Real,
					thrust::device_vector> solvePoisson(nx1,nx1Ghost,x1Max-x1Min);

	quakins::MemSaveReorderCopy<Real,2,nTot> copy1({1,0},{nx1Tot,nv1});
	quakins::MemSaveReorderCopy<Real,2,nTot> copy2({1,0},{nv1,nx1Tot});

	std::ofstream rho_out("rho",std::ios::out);
	std::ofstream phi_out("phi",std::ios::out);


	std::cout << "main loop start." << std::endl;
	for (int step=0; step<100; step++) {
		
		timer.tick("step"+std::to_string(step));
		for (int ie=0; ie<10; ie++) {

			fbmSolverX1(electron.begin(),nx1Tot);

			copy1(electron.begin(),electron_buf.begin());
			cal_dens(electron_buf.begin(), dens_e.begin());

			solvePoisson(dens_e,potential);

			copy2(electron_buf.begin(),electron.begin());
		}
		timer.tock();
		rho_out << dens_e;
		phi_out << potential;
	}

	std::ofstream out("df",std::ios::out);
	out << electron << std::endl;
}




