#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <cmath>
#include <fstream>
#include "FreeStreamSolver.hpp"
#include "PoissonSolver1D.hpp"
#include "Timer.h"
#include "ReorderCopy.hpp"
#include "MemSaveReorderCopy.hpp"
#include "reorder_copy.h"
#include "DensityReducer.hpp"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

using Real = float;
using Complex = std::complex<Real>;


constexpr std::size_t nx1 = 500;
constexpr std::size_t nv1 = 256;
constexpr std::size_t nx1Ghost = 6;
constexpr std::size_t nx1Tot = nx1Ghost*2+nx1;
constexpr std::size_t nTot = nx1Tot*nv1;

constexpr Real x1Max =  20;
constexpr Real x1Min =  0;
constexpr Real v1Max =  6;
constexpr Real v1Min = -6;

constexpr Real dt = 0.01;

template<typename T, 
		template<typename...> typename Container>
concept isAcontainer = requires (Container<T>& a) {
	a.begin(); a.end();
};

template<typename T, 
		template<typename...> typename Container>
requires isAcontainer<T,Container>
std::ostream& operator<<(std::ostream& os, 
		const Container<T>& vec) {
	thrust::copy(vec.begin(),vec.end(),
		std::ostream_iterator<T>(os," "));
	return os;
}

int main(int argc, char* argv[]) {

	Timer timer;
	
	quakins::CoordinateSystemHost<Real,2>
					_coord({nx1,nv1}, {nx1Ghost,0},
								 {x1Min,x1Max, v1Min,v1Max});
	quakins::WignerFunctionHost<Real,2> 
			_wf({nx1Tot,nv1});

	auto f = [](std::array<Real,2> z) {

		auto fx = [](Real x1) {
			return 1.+.1*std::cos(2.*M_PI/x1Max*x1);
		};
		auto fv = [](Real v1) {
			return std::exp(-std::pow(v1,2)/2.)/std::sqrt(2.*M_PI);
		};

		return fx(z[0])*fv(z[1]);
	};

	timer.tick("initializing...");
	quakins::init(_coord,_wf,f); timer.tock();
	
	std::ofstream bout("dfbegin",std::ios::out);
	bout << _wf.hVec << std::endl;

	quakins::fbm::FreeStreamSolver<Real,2,0> 
					fbmSolverX1(_wf,_coord,dt*.5);	

	
	thrust::device_vector<Real> 
		ion(_wf.nTot), ion_buf(_wf.nTot),
		electron(_wf.nTot), electron_buf(_wf.nTot);

	thrust::device_vector<Real> 
		dens_e(nx1Tot), dens_i(nx1Tot), potential(nx1Tot);

	quakins::DensityReducer<Real,nv1,nx1Tot,
					thrust::device_vector> cal_dens(v1Min,v1Max);

	quakins::FFTPoissonSolver1D<Real,
					thrust::device_vector> solvePoisson(nx1,nx1Ghost,x1Max-x1Min);

	quakins::MemSaveReorderCopy<Real,2,nTot> copy1({1,0},{nx1Tot,nv1});
	quakins::MemSaveReorderCopy<Real,2,nTot> copy2({1,0},{nv1,nx1Tot});

	std::ofstream rho_out("rho",std::ios::out);
	std::ofstream phi_out("phi",std::ios::out);

	electron = _wf.hVec;

	std::cout << "main loop start." << std::endl;
	for (int step=0; step<100; step++) {
		
		timer.tick("step"+std::to_string(step));
		for (int ie=0; ie<10; ie++) {

			fbmSolverX1(electron.begin(),nx1Tot);

			copy1(electron.begin(),electron_buf.begin());
			cal_dens(electron_buf.begin(), dens_e.begin());

			solvePoisson(dens_e,potential);

			copy2(electron_buf.begin(),electron.begin());
		}
		timer.tock();
		rho_out << dens_e;
		phi_out << potential;
	}

	std::ofstream out("df",std::ios::out);
	out << electron << std::endl;
}




