#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <cmath>
#include <fstream>
#include "FreeStreamSolver.hpp"
#include "Timer.h"
#include "ReorderCopy.hpp"
#include "reorder_copy.h"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

using Real = float;
using Complex = std::complex<Real>;
using VecH = thrust::host_vector<Real>;
using VecD = thrust::device_vector<Real>;


constexpr std::size_t nx1 = 508;
constexpr std::size_t nv1 = 256;
constexpr std::size_t nx1Ghost = 2;
constexpr std::size_t nx1Tot = nx1Ghost*2+nx1;

constexpr Real x1Max =  20;
constexpr Real x1Min =  0;
constexpr Real v1Max =  6;
constexpr Real v1Min = -6;

constexpr Real dt = 0.01;

template<typename T>
std::ostream& operator<<(std::ostream& os, 
		const thrust::host_vector<T>& vec) {
	thrust::copy(vec.begin(),vec.end(),
		std::ostream_iterator<T>(os," "));
	return os;
}

int main(int argc, char* argv[]) {

	Timer timer;
	
	quakins::CoordinateSystemHost<Real,2>
					_coord({nx1,nv1}, {nx1Ghost,0},
								 {x1Min,x1Max, v1Min,v1Max});
	quakins::WignerFunctionHost<Real,2> 
			_wf({nx1Tot,nv1});

	auto f = [](std::array<Real,2> z) {

		auto fx = [](Real x1) {
			return std::exp(-std::pow(x1-3,2));
		};
		auto fv = [](Real v1) {
			return std::exp(-std::pow(v1-2,2)/2.);
		};

		return fx(z[0])*fv(z[1]);
	};
	timer.tick("initializing...");
	quakins::init(_coord,_wf,f); timer.tock();
	
	std::ofstream bout("dfbegin",std::ios::out);
	bout << _wf.hVec << std::endl;

	quakins::fbm::FreeStreamSolver<Real,2,0> 
					fbmSolverX1(_wf,_coord,dt*.5);	


	timer.tick("requst memory on GPU...");
	thrust::device_vector<Real> test1(_wf.nTot);
	timer.tock(); 

	timer.tick("creating host reorder copy...");
	quakins::ReorderCopy<Real,2, true,
				thrust::host_vector> copy_h2d(_wf.N,{0,1});
	quakins::ReorderCopy<Real,2, false, 
				thrust::host_vector> copy_d2h
								({nx1Tot,nv1},{0,1});
	timer.tock();
	
		
	timer.tick("creating device reorder copy...");
	quakins::ReorderCopy<Real,2, true,
					thrust::device_vector> copy_d2d
									({nx1Tot,nv1},{0,1});
	timer.tock();

	copy_h2d(_wf.begin(), test1.begin());	
	std::cout << "main loop start." << std::endl;

	for (int step=0; step<300; step++) {
	//		timer.tick("step"+std::to_string(step));	
		fbmSolverX1(test1.begin(),nx1Tot);
	//	timer.tock();
	}

	copy_d2h(test1.begin(),_wf.begin());
	std::ofstream out("df",std::ios::out);
	out << _wf.hVec << std::endl;
	
}


