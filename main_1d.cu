#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <cmath>
#include <fstream>
#include "FreeStreamSolver.hpp"
#include "PoissonSolver1D.hpp"
#include "Timer.h"
#include "ReorderCopy.hpp"
#include "reorder_copy.h"
#include "DensityReducer.hpp"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

using Real = float;
using Complex = std::complex<Real>;
using VecH = thrust::host_vector<Real>;
using VecD = thrust::device_vector<Real>;


constexpr std::size_t nx1 = 508;
constexpr std::size_t nv1 = 256;
constexpr std::size_t nx1Ghost = 2;
constexpr std::size_t nx1Tot = nx1Ghost*2+nx1;

constexpr Real x1Max =  20;
constexpr Real x1Min =  0;
constexpr Real v1Max =  6;
constexpr Real v1Min = -6;

constexpr Real dt = 0.01;

template<typename T, 
		template<typename...> typename Container>
std::ostream& operator<<(std::ostream& os, 
		const Container<T>& vec) {
	thrust::copy(vec.begin(),vec.end(),
		std::ostream_iterator<T>(os," "));
	return os;
}

int main(int argc, char* argv[]) {

	Timer timer;
	
	quakins::CoordinateSystemHost<Real,2>
					_coord({nx1,nv1}, {nx1Ghost,0},
								 {x1Min,x1Max, v1Min,v1Max});
	quakins::WignerFunctionHost<Real,2> 
			_wf({nx1Tot,nv1});

	auto f = [](std::array<Real,2> z) {

		auto fx = [](Real x1) {
			return 1.+.1*std::cos(2.*M_PI/x1Max*x1);
		};
		auto fv = [](Real v1) {
			return std::exp(-std::pow(v1,2)/2.)/std::sqrt(2.*M_PI);
		};

		return fx(z[0])*fv(z[1]);
	};
	timer.tick("initializing...");
	quakins::init(_coord,_wf,f); timer.tock();
	
	std::ofstream bout("dfbegin",std::ios::out);
	bout << _wf.hVec << std::endl;

	quakins::fbm::FreeStreamSolver<Real,2,0> 
					fbmSolverX1(_wf,_coord,dt*.5);	

	timer.tick("creating host reorder copy...");
	quakins::ReorderCopy<Real,2, true,
				thrust::host_vector> copy_h2d(_wf.N,{0,1});
	quakins::ReorderCopy<Real,2, false, 
				thrust::host_vector> copy_d2h
								({nx1Tot,nv1},{0,1});
	timer.tock();
	
		
	timer.tick("creating device reorder copy...");
	quakins::ReorderCopy<Real,2, true,
					thrust::device_vector> copy_d2d_1
									({nx1Tot,nv1},{1,0});
	quakins::ReorderCopy<Real,2, true,
					thrust::device_vector> copy_d2d_2
									({nv1,nx1Tot},{1,0});
	timer.tock();
	
	thrust::device_vector<Real> test1(_wf.nTot);
	thrust::device_vector<Real> test2(_wf.nTot);
	copy_h2d(_wf.begin(),test1.begin());

	thrust::device_vector<Real> dens_e(nx1Tot), dens_i(nx1Tot);

	quakins::DensityReducer<Real,nv1,nx1Tot,
					thrust::device_vector> cal_dens(v1Min,v1Max);

	std::ofstream rho_out("rho",std::ios::out);

	std::cout << "main loop start." 
	<< std::endl; for (int step=0; step<500; step++) {

		timer.tick("step"+std::to_string(step));
		fbmSolverX1(test1.begin(),nx1Tot);
		copy_d2d_1(test1.begin(),test2.begin());

		cal_dens(test2.begin(), dens_e.begin());

		rho_out << dens_e;

		copy_d2d_2(test2.begin(),test1.begin());
		timer.tock();
	}

	std::ofstream out("df",std::ios::out);
	thrust::copy(test1.begin(), test1.end(),
							std::ostream_iterator<Real>(out," "));
	out << std::endl;
	
}







