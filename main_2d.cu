#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <cmath>
#include <fstream>
#include "FreeStreamSolver.hpp"
#include "Timer.h"
#include "MemSaveReorderCopy.hpp"
#include "PhaseSpaceInitialization.hpp"
#include "DensityReducer.hpp"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

using Real = float;
using Complex = std::complex<Real>;

constexpr std::size_t DIM = 4;

constexpr std::size_t nx1 = 100;
constexpr std::size_t nx2 = 80;
constexpr std::size_t nv1 = 66;
constexpr std::size_t nv2 = 60;
constexpr std::size_t nx1Ghost = 4;
constexpr std::size_t nx2Ghost = 4;
constexpr std::size_t nx1Tot = nx1Ghost*2+nx1;
constexpr std::size_t nx2Tot = nx2Ghost*2+nx2;
constexpr std::size_t nTot = nx1Tot*nx2Tot*nv1*nv2;

constexpr Real x1Max =  20, x2Max =  20;
constexpr Real x1Min =  0,  x2Min =  0;
constexpr Real v1Max =  6,  v2Max =  6;
constexpr Real v1Min = -6,  v2Min = -6;

constexpr Real dt = 0.01;


int main(int argc, char* argv[]) {

	Timer timer;

	timer.tick("Asking for GPU memory...");
	hipSetDevice(1);

	hipDeviceSetLimit(
		hipLimitMallocHeapSize, 1048576ULL*1024*3);
	hipDeviceSetLimit(
		hipLimitStackSize, 1048576ULL*1024*3);
	hipDeviceSetLimit(
		hipLimitPrintfFifoSize, 1048576ULL*1024*3);
	timer.tock();
	
	timer.tick("quakins start...");
	quakins::CoordinateSystem<Real,DIM>
					_coord({nx1,nx2,nv1,nv2},
								 {nx1Ghost,nx2Ghost,0,0},
								 {x1Min,x1Max,x2Min,x2Max,
								  v1Min,v1Max,v2Min,v2Max});

	auto f = [](std::array<Real,DIM> z) {

		auto fx = [](Real x1, Real x2) {
			return std::exp(-std::pow(x1-3,2)
							-std::pow(x2-10,2));	
		};
		auto fv = [](Real v1, Real v2) {
			return std::exp(-std::pow(v1+2,2)/2.
						-std::pow(v2,2)/1.);	
		};

		return fx(z[0],z[1])*fv(z[2],z[3]);
	};
	

	quakins::fbm::FreeStreamSolver<Real,DIM,0> 
					fbmSolverX1(_coord,dt*.5);	
	quakins::fbm::FreeStreamSolver<Real,DIM,1> 
					fbmSolverX2(_coord,dt*.5);

	quakins::MemSaveReorderCopy<Real,DIM,nTot>
					copy0({0,1,3,2},{nx1Tot,nx2Tot,nv1,nv2});
	quakins::MemSaveReorderCopy<Real,DIM,nTot>
					copy1({1,0,3,2},{nx1Tot,nx2Tot,nv2,nv1});
	quakins::MemSaveReorderCopy<Real,DIM,nTot>
					copy2({2,3,1,0},{nx2Tot,nx1Tot,nv1,nv2});
	quakins::MemSaveReorderCopy<Real,DIM,nTot>
					copy3({2,3,1,0},{nv1,nv2,nx1Tot,nx2Tot});


	thrust::device_vector<Real> test1(nTot), test2(nTot);
	thrust::device_vector<Real> dens_e(nx1Tot*nx2Tot), 
															dens_e_buf(nx1Tot*nx2Tot*nv2);

	quakins::DensityReducer<Real,nv1,nx1Tot*nx2Tot*nv2,
		thrust::device_vector> cal_dens_1(v1Min,v1Max);
	quakins::DensityReducer<Real,nv2,nx1Tot*nx2Tot,
		thrust::device_vector> cal_dens_2(v2Min,v2Max);


	timer.tock(); /* quakins start... */

	timer.tick("Phase space initialization...");
	quakins::PhaseSpaceInitialization<Real,DIM> init(&_coord);
	init(test2.begin(),f);
	timer.tock();

	copy0(test2.begin(),test1.begin());
	
	std::ofstream rho_out("rho",std::ios::out);

	std::cout << "main loop start." << std::endl;
	for (std::size_t step=0; step<400; step++) {
		timer.tick("step"+std::to_string(step));	

		fbmSolverX1(test1.begin(),nx1Tot*nx2Tot*nv2);
		
		copy1(test1.begin(),test2.begin());
	
		fbmSolverX2(test2.begin(),nx1Tot*nx2Tot*nv1);
		
		copy2(test2.begin(),test1.begin());
		
		cal_dens_1(test1.begin(),dens_e_buf.begin());
		cal_dens_2(dens_e_buf.begin(),dens_e.begin());
		
		if (step%10==0)
			rho_out << dens_e << std::endl;

		copy3(test1.begin(),test2.begin());
		test1 = test2;
		timer.tock();
	}

	std::ofstream out("df",std::ios::out);
	out << test1 << std::endl;
	
}


