#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <cmath>
#include <fstream>
#include "FreeStreamSolver.hpp"
#include "Timer.h"
#include "MemSaveReorderCopy.hpp"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

using Real = double;
using Complex = std::complex<Real>;
using VecH = thrust::host_vector<Real>;
using VecD = thrust::device_vector<Real>;


constexpr std::size_t nx1 = 220;
constexpr std::size_t nx2 = 206;
constexpr std::size_t nv1 = 82;
constexpr std::size_t nv2 = 80;
constexpr std::size_t nx1Ghost = 4;
constexpr std::size_t nx2Ghost = 2;
constexpr std::size_t nx1Tot = nx1Ghost*2+nx1;
constexpr std::size_t nx2Tot = nx2Ghost*2+nx2;
constexpr std::size_t nTot = nx1Tot*nx2Tot*nv1*nv2;

constexpr Real x1Max =  20, x2Max =  20;
constexpr Real x1Min =  0,  x2Min =  0;
constexpr Real v1Max =  6,  v2Max =  6;
constexpr Real v1Min = -6,  v2Min = -6;

constexpr Real dt = 0.01;

template<typename T, 
		template<typename...> typename Container>
concept isAcontainer = requires (Container<T>& a) {
	a.begin(); a.end();
};

template<typename T, 
		template<typename...> typename Container>
requires isAcontainer<T,Container>
std::ostream& operator<<(std::ostream& os, 
		const Container<T>& vec) {
	thrust::copy(vec.begin(),vec.end(),
		std::ostream_iterator<T>(os," "));
	return os;
}


int main(int argc, char* argv[]) {
	hipSetDevice(1);

	hipError_t err = hipDeviceSetLimit(
					hipLimitMallocHeapSize, 1048576ULL*1024*3);
	err = hipDeviceSetLimit(
					hipLimitStackSize, 1048576ULL*1024*4);
	err = hipDeviceSetLimit(
					hipLimitPrintfFifoSize, 1048576ULL*1024*1);
		

	Timer timer;
	
	quakins::CoordinateSystemHost<Real,4>
					_coord({nx1,nx2,nv1,nv2},
								 {nx1Ghost,nx2Ghost,0,0},
								 {x1Min,x1Max,x2Min,x2Max,
								  v1Min,v1Max,v2Min,v2Max});
	quakins::WignerFunctionHost<Real,4> 
			_wf({nx1Tot,nx2Tot,nv1,nv2});

	auto f = [](std::array<Real,4> z) {

		auto fx = [](Real x1, Real x2) {
			return std::exp(-std::pow(x1-3,2)
							-std::pow(x2-10,2));	
		};
		auto fv = [](Real v1, Real v2) {
			return std::exp(-std::pow(v1+2,2)/2.
						-std::pow(v2,2)/1.);	
		};

		return fx(z[0],z[1])*fv(z[2],z[3]);
	};
	timer.tick("initializing...");
	quakins::init(_coord,_wf,f); timer.tock();
	quakins::fbm::FreeStreamSolver<Real,4,0> 
					fbmSolverX1(_wf,_coord,dt*.5);	
	quakins::fbm::FreeStreamSolver<Real,4,1> 
					fbmSolverX2(_wf,_coord,dt*.5);	

		
	quakins::MemSaveReorderCopy<Real,4,nTot>
					copy0({0,1,3,2},{nx1Tot,nx2Tot,nv1,nv2});
	quakins::MemSaveReorderCopy<Real,4,nTot>
					copy1({1,0,3,2},{nx1Tot,nx2Tot,nv2,nv1});
	quakins::MemSaveReorderCopy<Real,4,nTot>
					copy2({1,0,3,2},{nx2Tot,nx1Tot,nv1,nv2});

	thrust::device_vector<Real> test1(nTot), test2(nTot);
	test2 = _wf.hVec;

	copy0(test2.begin(),test1.begin());
	
	std::ofstream bout("dfbegin",std::ios::out);
	//bout << test2 << std::endl;


	std::cout << "main loop start." << std::endl;
	for (int step=0; step<800; step++) {
		timer.tick("step"+std::to_string(step));	

		fbmSolverX1(test1.begin(),nx1Tot*nx2Tot*nv2);
		
		copy1(test1.begin(),test2.begin());
	
		fbmSolverX2(test2.begin(),nx1Tot*nx2Tot*nv1);
		
		copy2(test2.begin(),test1.begin());
		
		timer.tock();
	}

	std::ofstream out("df",std::ios::out);
	out << test1 << std::endl;
	
}


