#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <cmath>
#include <fstream>
#include "FreeStreamSolver.hpp"
#include "Timer.h"
#include "MemSaveReorderCopy.hpp"
#include "PhaseSpaceInitialization.hpp"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

using Real = float;
using Complex = std::complex<Real>;


constexpr std::size_t nx1 = 220;
constexpr std::size_t nx2 = 206;
constexpr std::size_t nv1 = 82;
constexpr std::size_t nv2 = 80;
constexpr std::size_t nx1Ghost = 4;
constexpr std::size_t nx2Ghost = 2;
constexpr std::size_t nx1Tot = nx1Ghost*2+nx1;
constexpr std::size_t nx2Tot = nx2Ghost*2+nx2;
constexpr std::size_t nTot = nx1Tot*nx2Tot*nv1*nv2;

constexpr Real x1Max =  20, x2Max =  20;
constexpr Real x1Min =  0,  x2Min =  0;
constexpr Real v1Max =  6,  v2Max =  6;
constexpr Real v1Min = -6,  v2Min = -6;

constexpr Real dt = 0.01;


int main(int argc, char* argv[]) {

	Timer timer;

	timer.tick("Asking for GPU memory...");
	hipSetDevice(1);

	hipDeviceSetLimit(
		hipLimitMallocHeapSize, 1048576ULL*1024*3);
	hipDeviceSetLimit(
		hipLimitStackSize, 1048576ULL*1024*3);
	hipDeviceSetLimit(
		hipLimitPrintfFifoSize, 1048576ULL*1024*3);
	timer.tock();
	
	timer.tick("quakins start...");
	quakins::CoordinateSystem<Real,4>
					_coord({nx1,nx2,nv1,nv2},
								 {nx1Ghost,nx2Ghost,0,0},
								 {x1Min,x1Max,x2Min,x2Max,
								  v1Min,v1Max,v2Min,v2Max});

	auto f = [](std::array<Real,4> z) {

		auto fx = [](Real x1, Real x2) {
			return std::exp(-std::pow(x1-3,2)
							-std::pow(x2-10,2));	
		};
		auto fv = [](Real v1, Real v2) {
			return std::exp(-std::pow(v1+2,2)/2.
						-std::pow(v2,2)/1.);	
		};

		return fx(z[0],z[1])*fv(z[2],z[3]);
	};
	

	quakins::fbm::FreeStreamSolver<Real,4,0> 
					fbmSolverX1(_coord,dt*.5);	
	quakins::fbm::FreeStreamSolver<Real,4,1> 
					fbmSolverX2(_coord,dt*.5);

	quakins::MemSaveReorderCopy<Real,4,nTot>
					copy0({0,1,3,2},{nx1Tot,nx2Tot,nv1,nv2});
	quakins::MemSaveReorderCopy<Real,4,nTot>
					copy1({1,0,3,2},{nx1Tot,nx2Tot,nv2,nv1});
	quakins::MemSaveReorderCopy<Real,4,nTot>
					copy2({1,0,3,2},{nx2Tot,nx1Tot,nv1,nv2});

	thrust::device_vector<Real> test1(nTot), test2(nTot);

	timer.tock(); /* quakins start... */

	timer.tick("Phase space initialization...");
	quakins::PhaseSpaceInitialization<Real,4> init(&_coord);
	init(test2.begin(),f);
	timer.tock();

	copy0(test2.begin(),test1.begin());
	
	std::ofstream bout("dfbegin",std::ios::out);
	bout << test2 << std::endl;


	std::cout << "main loop start." << std::endl;
	for (int step=0; step<800; step++) {
		timer.tick("step"+std::to_string(step));	

		fbmSolverX1(test1.begin(),nx1Tot*nx2Tot*nv2);
		
		copy1(test1.begin(),test2.begin());
	
		fbmSolverX2(test2.begin(),nx1Tot*nx2Tot*nv1);
		
		copy2(test2.begin(),test1.begin());
		
		timer.tock();
	}

	std::ofstream out("df",std::ios::out);
	out << test1 << std::endl;
	
}


