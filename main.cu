#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <cmath>
#include <fstream>
#include "FreeStreamSolver.hpp"
#include "Timer.h"
#include "ReorderCopy.hpp"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

using Real = float;
using Complex = std::complex<Real>;
using VecH = thrust::host_vector<Real>;
using VecD = thrust::device_vector<Real>;


constexpr std::size_t nx1 = 56;
constexpr std::size_t nx2 = 50;
constexpr std::size_t nv1 = 42;
constexpr std::size_t nv2 = 40;
constexpr std::size_t nx1Ghost = 2;
constexpr std::size_t nx2Ghost = 2;
constexpr std::size_t nx1Tot = nx1Ghost*2+nx1;
constexpr std::size_t nx2Tot = nx2Ghost*2+nx2;

constexpr Real x1Max =  20, x2Max =  20;
constexpr Real x1Min =  0,  x2Min =  0;
constexpr Real v1Max =  6,  v2Max =  6;
constexpr Real v1Min = -6,  v2Min = -6;

constexpr Real dt = 0.01;

template<typename T>
std::ostream& operator<<(std::ostream& os, 
		const thrust::host_vector<T>& vec) {
	thrust::copy(vec.begin(),vec.end(),
		std::ostream_iterator<T>(os," "));
	return os;
}

int main(int argc, char* argv[]) {

	Timer timer;
	
	quakins::CoordinateSystemHost<Real,4>
					_coord({nx1,nx2,nv1,nv2},
								 {nx1Ghost,nx2Ghost,0,0},
								 {x1Min,x1Max,x2Min,x2Max,
								  v1Min,v1Max,v2Min,v2Max});
	quakins::WignerFunctionHost<Real,4> 
			_wf({nx1Tot,nx2Tot,nv1,nv2});

	auto f = [](std::array<Real,4> z) {

		auto fx = [](Real x1, Real x2) {
			return std::exp(-std::pow(x1-3,2)
							-std::pow(x2-10,2));	
		};
		auto fv = [](Real v1, Real v2) {
			return std::exp(-std::pow(v1-2,2)/2.
						-std::pow(v2,2)/2.);	
		};

		return fx(z[0],z[1])*fv(z[2],z[3]);
	};
	timer.tick("initializing...");
	quakins::init(_coord,_wf,f); timer.tock();
	
	std::ofstream bout("dfbegin",std::ios::out);
	bout << _wf.hVec << std::endl;

	quakins::fbm::FreeStreamSolver<Real,4,0> 
					fbmSolverX1(_wf,_coord,dt*.5);	
	quakins::fbm::FreeStreamSolver<Real,4,1> 
					fbmSolverX2(_wf,_coord,dt*.5);	


	timer.tick("requst memory on GPU...");
	thrust::device_vector<Real> test1(_wf.nTot);
	thrust::device_vector<Real> test2(_wf.nTot);
	timer.tock(); 

	timer.tick("creating host reorder copy...");
	quakins::ReorderCopy<Real,4, 
					thrust::host_vector> copy_h2d(_wf.N,{0,1,3,2});
	timer.tock();
	
	timer.tick("transfering data from host to device...");
	copy_h2d(_wf.begin(),test1.begin());
	timer.tock();
		
	timer.tick("creating device reorder copy...");
	quakins::ReorderCopy<Real,4, 
					thrust::device_vector> copy_d2d_1
									({nx1Tot,nx2Tot,nv2,nv1},{1,0,3,2});
	quakins::ReorderCopy<Real,4, 
					thrust::device_vector> copy_d2d_2
									({nx2Tot,nx1Tot,nv1,nv2},{1,0,3,2});
	timer.tock();


	std::cout << "main loop start." << std::endl;
	for (int step=0; step<300; step++) {
		timer.tick("step"+std::to_string(step));	
		fbmSolverX1(test1.begin(),nx1Tot*nx2Tot*nv2);
		copy_d2d_1(test1.begin(),test2.begin());
		fbmSolverX2(test2.begin(),nx1Tot*nx2Tot*nv1);
		copy_d2d_2(test2.begin(),test1.begin());
		timer.tock();
	}
	
	timer.tick("transfering data from device to host...");
  _wf.hVec = test1;
	timer.tock();

	std::ofstream out("df",std::ios::out);
	out << _wf.hVec << std::endl;
	
}


